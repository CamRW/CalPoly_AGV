#include "hip/hip_runtime.h"
/*
Copyright 2022 NVIDIA CORPORATION

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "nvblox/core/internal/cuda/atomic_float.cuh"
#include "nvblox/geometry/bounding_boxes.h"
#include "nvblox/geometry/bounding_spheres.h"
#include "nvblox/gpu_hash/internal/cuda/gpu_hash_interface.cuh"
#include "nvblox/gpu_hash/internal/cuda/gpu_indexing.cuh"
#include "nvblox/gpu_hash/internal/cuda/gpu_set.cuh"
#include "nvblox/utils/timing.h"

#include "nvblox/integrators/esdf_integrator.h"

namespace nvblox {

struct TsdfSiteFunctor {
  __device__ bool isVoxelObserved(const TsdfVoxel& tsdf_voxel) const {
    return tsdf_voxel.weight >= min_weight;
  }

  __device__ bool isVoxelInsideObject(const TsdfVoxel& tsdf_voxel) const {
    return tsdf_voxel.distance <= 0.0f;
  }

  __device__ bool isVoxelNearSurface(const TsdfVoxel& tsdf_voxel) const {
    return fabsf(tsdf_voxel.distance) <= max_site_distance_m;
  }

  __device__ void updateSquashedExtremumAtomic(const TsdfVoxel& tsdf_voxel,
                                               TsdfVoxel* current_value) const {
    atomicMinFloat(&current_value->distance, tsdf_voxel.distance);
  }

  float min_weight;
  float max_site_distance_m;
};

struct OccupancySiteFunctor {
  __device__ bool isVoxelObserved(const OccupancyVoxel& occupancy_voxel) const {
    constexpr float kEps = 1e-4;
    constexpr float kLogOddsZeroPointFive = 0;
    return fabsf(occupancy_voxel.log_odds - kLogOddsZeroPointFive) > kEps;
  }

  __device__ bool isVoxelInsideObject(
      const OccupancyVoxel& occupancy_voxel) const {
    return occupancy_voxel.log_odds > occupied_threshold_log_odds;
  }

  __device__ bool isVoxelNearSurface(
      const OccupancyVoxel& occupancy_voxel) const {
    return true;
  }

  __device__ void updateSquashedExtremumAtomic(
      const OccupancyVoxel& occupancy_voxel,
      OccupancyVoxel* current_voxel) const {
    atomicMaxFloat(&current_voxel->log_odds, occupancy_voxel.log_odds);
  }

  float occupied_threshold_log_odds;
};

EsdfIntegrator::~EsdfIntegrator() {
  if (cuda_stream_ != nullptr) {
    hipStreamDestroy(cuda_stream_);
  }
}

float EsdfIntegrator::max_distance_m() const { return max_distance_m_; }

float EsdfIntegrator::max_site_distance_vox() const {
  return tsdf_max_site_distance_vox_;
}

float EsdfIntegrator::min_weight() const { return tsdf_min_weight_; }

void EsdfIntegrator::max_distance_m(float max_distance_m) {
  CHECK_GT(max_distance_m, 0.0f);
  max_distance_m_ = max_distance_m;
}

void EsdfIntegrator::max_site_distance_vox(float max_site_distance_vox) {
  CHECK_GT(max_site_distance_vox, 0.0f);
  tsdf_max_site_distance_vox_ = max_site_distance_vox;
}

void EsdfIntegrator::min_weight(float min_weight) {
  CHECK_GT(min_weight, 0.0f);
  tsdf_min_weight_ = min_weight;
}

float EsdfIntegrator::occupied_threshold() const {
  return probabilityFromLogOdds(occupied_threshold_log_odds_);
}

void EsdfIntegrator::occupied_threshold(float occupied_threshold) {
  CHECK_GE(occupied_threshold, 0.0f);
  CHECK_LE(occupied_threshold, 1.0f);
  occupied_threshold_log_odds_ = logOddsFromProbability(occupied_threshold);
}

// Integrate the entire layer.
void EsdfIntegrator::integrateLayer(const TsdfLayer& tsdf_layer,
                                    EsdfLayer* esdf_layer) {
  std::vector<Index3D> block_indices = tsdf_layer.getAllBlockIndices();

  integrateBlocks(tsdf_layer, block_indices, esdf_layer);
}

template <typename LayerType>
void EsdfIntegrator::integrateBlocks(const LayerType& layer,
                                     const std::vector<Index3D>& block_indices,
                                     EsdfLayer* esdf_layer) {
  timing::Timer esdf_timer("esdf/integrate");

  if (block_indices.empty()) {
    return;
  }

  // First, check if the stream exists. If not, create one.
  if (cuda_stream_ == nullptr) {
    checkCudaErrors(hipStreamCreate(&cuda_stream_));
  }

  timing::Timer allocate_timer("esdf/integrate/allocate");
  // First, allocate all the destination blocks.
  allocateBlocksOnCPU(block_indices, esdf_layer);
  allocate_timer.Stop();

  timing::Timer mark_timer("esdf/integrate/mark_sites");
  // Then, mark all the sites on GPU.
  // This finds all the blocks that are eligible to be parents.
  markAllSitesCombined(layer, block_indices, esdf_layer,
                       &updated_indices_device_, &to_clear_indices_device_);
  mark_timer.Stop();

  if (!to_clear_indices_device_.empty()) {
    timing::Timer compute_timer("esdf/integrate/clear");
    clearAllInvalid(to_clear_indices_device_.toVector(), esdf_layer,
                    &cleared_block_indices_device_);
  }

  timing::Timer compute_timer("esdf/integrate/compute");
  // Parallel block banding on GPU.
  computeEsdfCombined(updated_indices_device_, esdf_layer);
  if (!cleared_block_indices_device_.empty()) {
    computeEsdfCombined(cleared_block_indices_device_, esdf_layer);
  }
  compute_timer.Stop();
}

void EsdfIntegrator::integrateBlocks(const TsdfLayer& tsdf_layer,
                                     const std::vector<Index3D>& block_indices,
                                     EsdfLayer* esdf_layer) {
  integrateBlocks<TsdfLayer>(tsdf_layer, block_indices, esdf_layer);
}

void EsdfIntegrator::integrateBlocks(const OccupancyLayer& occupancy_layer,
                                     const std::vector<Index3D>& block_indices,
                                     EsdfLayer* esdf_layer) {
  integrateBlocks<OccupancyLayer>(occupancy_layer, block_indices, esdf_layer);
}

template <typename LayerType>
void EsdfIntegrator::integrateSlice(const LayerType& layer,
                                    const std::vector<Index3D>& block_indices,
                                    float z_min, float z_max, float z_output,
                                    EsdfLayer* esdf_layer) {
  timing::Timer esdf_timer("esdf/integrate_slice");

  if (block_indices.empty()) {
    return;
  }

  // First, check if the stream exists. If not, create one.
  if (cuda_stream_ == nullptr) {
    checkCudaErrors(hipStreamCreate(&cuda_stream_));
  }

  timing::Timer mark_timer("esdf/integrate_slice/mark_sites");
  // Then, mark all the sites on GPU.
  // This finds all the blocks that are eligible to be parents.
  markSitesInSliceCombined(layer, block_indices, z_min, z_max, z_output,
                           esdf_layer, &updated_indices_device_,
                           &to_clear_indices_device_);
  mark_timer.Stop();

  if (!to_clear_indices_device_.empty()) {
    timing::Timer compute_timer("esdf/integrate/clear");
    clearAllInvalid(to_clear_indices_device_.toVector(), esdf_layer,
                    &cleared_block_indices_device_);
  }

  timing::Timer compute_timer("esdf/integrate_slice/compute");
  // Parallel block banding on GPU.
  computeEsdfCombined(updated_indices_device_, esdf_layer);
  if (!cleared_block_indices_device_.empty()) {
    computeEsdfCombined(cleared_block_indices_device_, esdf_layer);
  }
  compute_timer.Stop();
}

void EsdfIntegrator::integrateSlice(const TsdfLayer& layer,
                                    const std::vector<Index3D>& block_indices,
                                    float z_min, float z_max, float z_output,
                                    EsdfLayer* esdf_layer) {
  integrateSlice<TsdfLayer>(layer, block_indices, z_min, z_max, z_output,
                            esdf_layer);
}

void EsdfIntegrator::integrateSlice(const OccupancyLayer& layer,
                                    const std::vector<Index3D>& block_indices,
                                    float z_min, float z_max, float z_output,
                                    EsdfLayer* esdf_layer) {
  integrateSlice<OccupancyLayer>(layer, block_indices, z_min, z_max, z_output,
                                 esdf_layer);
}

void EsdfIntegrator::allocateBlocksOnCPU(
    const std::vector<Index3D>& block_indices, EsdfLayer* esdf_layer) {
  // We want to allocate all ESDF layer blocks and copy over the sites.
  for (const Index3D& block_index : block_indices) {
    esdf_layer->allocateBlockAtIndex(block_index);
  }
}

__device__ void clearVoxelDevice(EsdfVoxel* voxel,
                                 float max_squared_distance_vox) {
  voxel->parent_direction.setZero();
  voxel->squared_distance_vox = max_squared_distance_vox;
  voxel->is_site = false;
}

// Mark sites to lower & clear.
// Block size MUST be voxels_per_side x voxels_per_side x voxel_per_size.
// Grid size can be anything.
template <typename BlockType, typename SiteFunctorType>
__global__ void markAllSitesCombinedKernel(
    int num_blocks, Index3D* block_indices,
    Index3DDeviceHashMapType<BlockType> input_layer_block_hash,
    Index3DDeviceHashMapType<EsdfBlock> esdf_block_hash,
    const SiteFunctorType site_functor, float max_squared_distance_vox,
    Index3D* updated_vec, int* updated_vec_size, Index3D* to_clear_vec,
    int* to_clear_vec_size) {
  dim3 voxel_index = threadIdx;
  int block_idx = blockIdx.x;

  using VoxelType = typename BlockType::VoxelType;

  __shared__ BlockType* block_ptr;
  __shared__ EsdfBlock* esdf_block;
  __shared__ int updated;
  __shared__ int to_clear;
  __syncthreads();

  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    block_ptr = nullptr;
    esdf_block = nullptr;
    updated = false;
    to_clear = false;
    auto tsdf_it = input_layer_block_hash.find(block_indices[block_idx]);
    if (tsdf_it != input_layer_block_hash.end()) {
      block_ptr = tsdf_it->second;
    }
    auto esdf_it = esdf_block_hash.find(block_indices[block_idx]);
    if (esdf_it != esdf_block_hash.end()) {
      esdf_block = esdf_it->second;
    }
  }
  __syncthreads();
  if (block_ptr == nullptr || esdf_block == nullptr) {
    return;
  }

  // Get the correct voxel for this index.
  const VoxelType* voxel_ptr =
      &block_ptr->voxels[voxel_index.x][voxel_index.y][voxel_index.z];
  EsdfVoxel* esdf_voxel =
      &esdf_block->voxels[voxel_index.x][voxel_index.y][voxel_index.z];
  if (site_functor.isVoxelObserved(*voxel_ptr)) {
    // Mark as inside if the voxel distance is negative.
    const bool is_inside = site_functor.isVoxelInsideObject(*voxel_ptr);
    if (esdf_voxel->is_inside && is_inside == false) {
      clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
      to_clear = true;
    }
    esdf_voxel->is_inside = is_inside;
    if (is_inside && site_functor.isVoxelNearSurface(*voxel_ptr)) {
      esdf_voxel->is_site = true;
      esdf_voxel->squared_distance_vox = 0.0f;
      esdf_voxel->parent_direction.setZero();
      updated = true;
    } else {
      if (esdf_voxel->is_site) {
        esdf_voxel->is_site = false;
        // This voxel needs to be cleared.
        clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
        to_clear = true;
      } else if (!esdf_voxel->observed) {
        // This is a brand new voxel.
        clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
      } else if (esdf_voxel->squared_distance_vox <= 1e-4) {
        // This is an invalid voxel that should be cleared.
        clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
        to_clear = true;
      }
    }
    esdf_voxel->observed = true;
  } else {
    clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
    to_clear = true;
    esdf_voxel->observed = false;
  }

  __syncthreads();

  if (threadIdx.x == 1 && threadIdx.y == 1 && threadIdx.z == 1) {
    if (updated) {
      updated_vec[atomicAdd(updated_vec_size, 1)] = block_indices[block_idx];
    }
    if (to_clear) {
      to_clear_vec[atomicAdd(to_clear_vec_size, 1)] = block_indices[block_idx];
    }
  }
}

template <typename T>
struct conditional_false : std::false_type {};

// NOTE(alexmillane): We use these types below to eliminate the default
// constructors of the voxel types such that they can be used with shared
// memory. In our testing this is no longer required after CUDA 11.8. At some
// point in the future (once the jetpack moves to >= 11.8) we can remove this.
namespace {

struct TsdfVoxelShared : public TsdfVoxel {
  TsdfVoxelShared() {}
};

struct OccupancyVoxelShared : public OccupancyVoxel {
  OccupancyVoxelShared() {}
};

template <typename VoxelType>
struct SharedVoxel;

template <>
struct SharedVoxel<TsdfVoxel> {
  typedef TsdfVoxelShared type;
};

template <>
struct SharedVoxel<OccupancyVoxel> {
  typedef OccupancyVoxelShared type;
};

}  // namespace

/// Thread size MUST be 8x8x8, block size can be anything.
template <typename BlockType, typename SiteFunctorType>
__global__ void markSitesInSliceCombinedKernel(
    int num_blocks, Index3D* block_indices,
    Index3DDeviceHashMapType<BlockType> input_layer_block_hash,
    Index3DDeviceHashMapType<EsdfBlock> esdf_block_hash,
    const SiteFunctorType site_functor, float max_squared_distance_vox,
    int output_voxel_index, float min_height_z, float max_height_z,
    float block_size, Index3D* updated_vec, int* updated_vec_size,
    Index3D* to_clear_vec, int* to_clear_vec_size) {
  dim3 voxel_index = threadIdx;
  voxel_index.z = output_voxel_index;
  int layer_index = threadIdx.z;
  int num_layers = blockDim.z;

  using VoxelType = typename BlockType::VoxelType;

  constexpr int kVoxelsPerSide = BlockType::kVoxelsPerSide;

  // First port-of-call is squashing a 3D band of the surface reconstruction to
  // 2D. First we allocating 2D arrays for the output.
  __shared__ bool observed[kVoxelsPerSide][kVoxelsPerSide];
  __shared__ typename SharedVoxel<VoxelType>::type voxel_slice[kVoxelsPerSide]
                                                              [kVoxelsPerSide];

  __shared__ Index3D min_block_index, min_voxel_index, max_block_index,
      max_voxel_index;
  __shared__ Index3D esdf_block_index;
  __shared__ EsdfBlock* esdf_block;
  __shared__ bool updated, cleared;

  // Initialize these.
  if (layer_index == 0) {
    observed[voxel_index.x][voxel_index.y] = false;
    if constexpr (std::is_same<TsdfVoxel, VoxelType>::value) {
      // NOTE(alexmillane): We don't use the weight in the slice, so we don't
      // initialize it.
      voxel_slice[voxel_index.x][voxel_index.y].distance =
          2 * max_squared_distance_vox;
    } else if constexpr (std::is_same<OccupancyVoxel, VoxelType>::value) {
      voxel_slice[voxel_index.x][voxel_index.y].log_odds = 0.0f;
    } else {
      static_assert(conditional_false<BlockType>::value,
                    "Slicing not specialized to LayerType yet.");
    }
  }
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    updated = false;
    cleared = false;
    getBlockAndVoxelIndexFromPositionInLayer(
        block_size, Vector3f(0.0f, 0.0f, min_height_z), &min_block_index,
        &min_voxel_index);
    getBlockAndVoxelIndexFromPositionInLayer(
        block_size, Vector3f(0.0f, 0.0f, max_height_z), &max_block_index,
        &max_voxel_index);
    esdf_block_index = block_indices[blockIdx.x];
    esdf_block = nullptr;
    auto it = esdf_block_hash.find(esdf_block_index);
    if (it != esdf_block_hash.end()) {
      esdf_block = it->second;
    }
  }
  __syncthreads();

  // This shouldn't happen.
  if (esdf_block == nullptr) {
    printf(
        "No output block exists in markSitesInSliceCombinedKernel(). Shouldn't "
        "happen.\n");
    return;
  }

  // Get the correct block for this.
  Index3D block_index = esdf_block_index;
  block_index.z() = min_block_index.z() + layer_index;

  const BlockType* block_ptr = nullptr;
  auto it = input_layer_block_hash.find(block_index);
  if (it != input_layer_block_hash.end()) {
    block_ptr = it->second;
  }

  // There's also null pointers in there.
  if (block_ptr != nullptr) {
    // Iterate over all of the voxels in this block.
    int start_index = 0;
    int end_index = kVoxelsPerSide;
    if (layer_index == 0) {
      start_index = min_voxel_index.z();
    }
    if (layer_index == num_layers - 1) {
      end_index = max_voxel_index.z();
    }
    for (int i = start_index; i < end_index; i++) {
      const VoxelType* voxel_ptr =
          &block_ptr->voxels[voxel_index.x][voxel_index.y][i];
      // Get the correct voxel for this index.
      if (site_functor.isVoxelObserved(*voxel_ptr)) {
        observed[voxel_index.x][voxel_index.y] = true;
        site_functor.updateSquashedExtremumAtomic(
            *voxel_ptr, &voxel_slice[voxel_index.x][voxel_index.y]);
      }
    }
  }

  // sync threads across everyone trying to update this voxel
  __syncthreads();

  // Ok now only if we're layer 0 do we compare the new and old values and
  // decide what to output.
  if (layer_index == 0) {
    EsdfVoxel* esdf_voxel =
        &esdf_block->voxels[voxel_index.x][voxel_index.y][voxel_index.z];

    // Case 0: Just skip it if it's unobserved. We don't care.
    if (observed[voxel_index.x][voxel_index.y]) {
      // Determine if the new value puts us inside or in a site.
      const bool is_inside = site_functor.isVoxelInsideObject(
          voxel_slice[voxel_index.x][voxel_index.y]);
      const bool is_site =
          is_inside && site_functor.isVoxelNearSurface(
                           voxel_slice[voxel_index.x][voxel_index.y]);

      // First handle the case where the voxel is a site.
      if (is_site) {
        if (esdf_voxel->is_site) {
          // Ok whatever. Add to the site list.
          // Its existing values are fine.
          updated = true;
        } else {
          // Wasn't a site before, is now.
          esdf_voxel->observed = true;
          esdf_voxel->squared_distance_vox = 0.0f;
          esdf_voxel->parent_direction.setZero();
          esdf_voxel->is_site = true;
          updated = true;
        }
      } else {
        // Here we have to double-check what's going on.
        // If it was a site before, and isn't anymore, we have to clear it.
        if (esdf_voxel->is_site) {
          esdf_voxel->is_site = false;
          clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
          cleared = true;
        }
        // Otherwise just leave it alone unless it's brand new.
        if (!esdf_voxel->observed) {
          esdf_voxel->observed = true;
          clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
        } else if (esdf_voxel->is_inside != is_inside) {
          // In case the sidedness swapped, clear the voxel.
          clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
          cleared = true;
        } else if (esdf_voxel->squared_distance_vox <= 0.0f) {
          // This is somehow invalidly marked as a site despite the fact
          // it shouldn't be.
          clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
          cleared = true;
        }
      }
      // Make the sidedness match.
      esdf_voxel->is_inside = is_inside;
    } else {
      clearVoxelDevice(esdf_voxel, max_squared_distance_vox);
      cleared = true;
      esdf_voxel->observed = false;
    }
  }

  // Now output the updated and cleared.
  __syncthreads();
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    if (updated) {
      updated_vec[atomicAdd(updated_vec_size, 1)] = esdf_block_index;
    }
    if (cleared) {
      to_clear_vec[atomicAdd(to_clear_vec_size, 1)] = esdf_block_index;
    }
  }
}

__device__ void sweepSingleBand(Index3D voxel_index, int sweep_axis,
                                float max_squared_distance_vox,
                                EsdfBlock* esdf_block) {
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  Index3D last_site;
  bool site_found;
  // Sweep sweep sweep.
  // First we sweep forward, then backwards.
  for (int i = 0; i < 2; i++) {
    last_site = Index3D::Zero();
    site_found = false;
    int direction = 1;
    int start_voxel = 0;
    int end_voxel = kVoxelsPerSide;
    if (i == 1) {
      direction = -1;
      start_voxel = kVoxelsPerSide - 1;
      end_voxel = -1;
    }

    for (voxel_index(sweep_axis) = start_voxel;
         voxel_index(sweep_axis) != end_voxel;
         voxel_index(sweep_axis) += direction) {
      EsdfVoxel* esdf_voxel =
          &esdf_block
               ->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];
      if (!esdf_voxel->observed) {
        continue;
      }
      // If this voxel is itself a site, then mark this for future voxels.
      if (esdf_voxel->is_site) {
        last_site = voxel_index;
        site_found = true;
      } else if (!site_found) {
        // If this voxel isn't a site but we haven't found a site yet,
        // then if this voxel is valid we set it as the site.
        if (esdf_voxel->squared_distance_vox < max_squared_distance_vox) {
          site_found = true;
          last_site = esdf_voxel->parent_direction + voxel_index;
        }
      } else {
        // If we've found the site, then should just decide what to do
        // here.
        Index3D potential_direction = last_site - voxel_index;
        float potential_distance = potential_direction.squaredNorm();
        // Either it hasn't been set at all or it's closer to the site
        // than to its current value.
        if (esdf_voxel->squared_distance_vox > potential_distance) {
          esdf_voxel->parent_direction = potential_direction;
          esdf_voxel->squared_distance_vox = potential_distance;
        } else if (esdf_voxel->squared_distance_vox <
                   max_squared_distance_vox) {
          // If the current value is a better site, then set it as a site.
          last_site = esdf_voxel->parent_direction + voxel_index;
        }
      }
    }
  }
}

__device__ bool updateSingleNeighbor(const EsdfBlock* esdf_block,
                                     const Index3D& voxel_index,
                                     const Index3D& neighbor_voxel_index,
                                     int axis, int direction,
                                     float max_squared_distance_vox,
                                     EsdfBlock* neighbor_block) {
  const EsdfVoxel* esdf_voxel =
      &esdf_block->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];
  EsdfVoxel* neighbor_voxel =
      &neighbor_block
           ->voxels[neighbor_voxel_index.x()][neighbor_voxel_index.y()]
                   [neighbor_voxel_index.z()];
  if (!esdf_voxel->observed || !neighbor_voxel->observed ||
      neighbor_voxel->is_site ||
      esdf_voxel->squared_distance_vox >= max_squared_distance_vox) {
    return false;
  }
  // Determine if we can update this.
  Eigen::Vector3i potential_direction = esdf_voxel->parent_direction;
  potential_direction(axis) -= direction;
  float potential_distance = potential_direction.squaredNorm();
  // TODO: might be some concurrency issues here, have to be a bit careful
  // on the corners/edges.
  if (neighbor_voxel->squared_distance_vox > potential_distance) {
    neighbor_voxel->parent_direction = potential_direction;
    neighbor_voxel->squared_distance_vox = potential_distance;
    return true;
  }
  return false;
}

__device__ bool clearSingleNeighbor(const EsdfBlock* esdf_block,
                                    const Index3D& voxel_index,
                                    const Index3D& neighbor_voxel_index,
                                    int axis, int direction,
                                    float max_squared_distance_vox,
                                    EsdfBlock* neighbor_block) {
  const EsdfVoxel* esdf_voxel =
      &esdf_block->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];
  EsdfVoxel* neighbor_voxel =
      &neighbor_block
           ->voxels[neighbor_voxel_index.x()][neighbor_voxel_index.y()]
                   [neighbor_voxel_index.z()];

  if (esdf_voxel->squared_distance_vox < max_squared_distance_vox ||
      !esdf_voxel->observed || neighbor_voxel->is_site ||
      neighbor_voxel->squared_distance_vox >= max_squared_distance_vox) {
    return false;
  }
  // Determine if we can update this.
  Index3D parent_voxel_dir = neighbor_voxel->parent_direction;
  if ((direction > 0 && parent_voxel_dir(axis) > 0) ||
      (direction < 0 && parent_voxel_dir(axis) < 0)) {
    return false;
  }

  clearVoxelDevice(neighbor_voxel, max_squared_distance_vox);
  return true;
}

OccupancySiteFunctor EsdfIntegrator::getSiteFunctor(
    const OccupancyLayer& layer) {
  OccupancySiteFunctor functor;
  functor.occupied_threshold_log_odds = occupied_threshold_log_odds_;
  return functor;
}

TsdfSiteFunctor EsdfIntegrator::getSiteFunctor(const TsdfLayer& layer) {
  TsdfSiteFunctor functor;
  functor.min_weight = tsdf_min_weight_;
  functor.max_site_distance_m =
      tsdf_max_site_distance_vox_ * layer.voxel_size();
  return functor;
}

template <typename LayerType>
void EsdfIntegrator::markAllSitesCombined(
    const LayerType& layer, const std::vector<Index3D>& block_indices,
    EsdfLayer* esdf_layer, device_vector<Index3D>* blocks_with_sites,
    device_vector<Index3D>* cleared_blocks) {
  CHECK_NOTNULL(esdf_layer);
  CHECK_NOTNULL(blocks_with_sites);

  if (block_indices.empty()) {
    return;
  }

  // Caching.
  const float voxel_size = layer.voxel_size();
  const float max_distance_vox = max_distance_m_ / voxel_size;
  const float max_squared_distance_vox = max_distance_vox * max_distance_vox;
  // Cache the minimum distance in metric size.
  const float max_site_distance_m = tsdf_max_site_distance_vox_ * voxel_size;

  int num_blocks = block_indices.size();

  block_indices_device_ = block_indices;
  blocks_with_sites->resize(num_blocks);
  cleared_blocks->resize(num_blocks);

  if (updated_counter_device_ == nullptr || updated_counter_host_ == nullptr) {
    updated_counter_device_ = make_unified<int>(MemoryType::kDevice);
    updated_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  updated_counter_device_.setZero();
  if (cleared_counter_device_ == nullptr || cleared_counter_host_ == nullptr) {
    cleared_counter_device_ = make_unified<int>(MemoryType::kDevice);
    cleared_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  cleared_counter_device_.setZero();

  GPULayerView<EsdfBlock> esdf_layer_view = esdf_layer->getGpuLayerView();
  GPULayerView<typename LayerType::BlockType> input_layer_view =
      layer.getGpuLayerView();

  // Get the marking functions for this layer type
  auto site_functor = getSiteFunctor(layer);

  // Call the kernel.
  int dim_block = num_blocks;
  constexpr int kVoxelsPerSide = EsdfBlock::kVoxelsPerSide;
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  // Call kernel, passing functor
  markAllSitesCombinedKernel<<<dim_block, dim_threads, 0, cuda_stream_>>>(
      num_blocks, block_indices_device_.data(),  // NOLINT
      input_layer_view.getHash().impl_,          // NOLINT
      esdf_layer_view.getHash().impl_,           // NOLINT
      site_functor,                              // NOLINT
      max_squared_distance_vox,                  // NOLINT
      blocks_with_sites->data(),                 // NOLINT
      updated_counter_device_.get(),             // NOLINT
      cleared_blocks->data(),                    // NOLINT
      cleared_counter_device_.get());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));
  checkCudaErrors(hipPeekAtLastError());

  timing::Timer pack_out_timer("esdf/integrate/mark_sites/pack_out");
  updated_counter_device_.copyTo(updated_counter_host_);
  cleared_counter_device_.copyTo(cleared_counter_host_);

  blocks_with_sites->resize(*updated_counter_host_);
  cleared_blocks->resize(*cleared_counter_host_);
  pack_out_timer.Stop();
}

template <typename LayerType>
void EsdfIntegrator::markSitesInSliceCombined(
    const LayerType& input_layer, const std::vector<Index3D>& block_indices,
    float min_z, float max_z, float output_z, EsdfLayer* esdf_layer,
    device_vector<Index3D>* updated_blocks,
    device_vector<Index3D>* cleared_blocks) {
  if (block_indices.empty()) {
    return;
  }

  // Caching.
  const float voxel_size = input_layer.voxel_size();
  const float max_distance_vox = max_distance_m_ / voxel_size;
  const float max_squared_distance_vox = max_distance_vox * max_distance_vox;
  // Cache the minimum distance in metric size.
  const float max_site_distance_m = tsdf_max_site_distance_vox_ * voxel_size;
  constexpr int kVoxelsPerSide = EsdfBlock::kVoxelsPerSide;

  // We are going to subsample the block_indices.
  // We need to figure out all the output blocks, which will be a subset
  // of the input blocks.

  // Ok first figure out how many layers we could have.
  Index3D min_block_index;
  Index3D min_voxel_index;
  getBlockAndVoxelIndexFromPositionInLayer(input_layer.block_size(),
                                           Vector3f(0.0f, 0.0f, min_z),
                                           &min_block_index, &min_voxel_index);
  const int min_block_index_z = min_block_index.z();
  const int min_voxel_index_z = min_voxel_index.z();
  Index3D max_block_index;
  Index3D max_voxel_index;
  getBlockAndVoxelIndexFromPositionInLayer(input_layer.block_size(),
                                           Vector3f(0.0f, 0.0f, max_z),
                                           &max_block_index, &max_voxel_index);
  const int max_block_index_z = max_block_index.z();
  const int max_voxel_index_z = max_voxel_index.z();

  // And figure out what the index of the output voxel is.
  // std::pair<Index3D, Index3D> output_block_and_voxel_index
  Index3D output_block_index;
  Index3D output_voxel_index;
  getBlockAndVoxelIndexFromPositionInLayer(
      input_layer.block_size(), Vector3f(0.0f, 0.0f, output_z),
      &output_block_index, &output_voxel_index);
  const int output_block_index_z = output_block_index.z();
  const int output_voxel_index_z = output_voxel_index.z();

  // There is always at least 1 layer.
  int num_vertical_layers = max_block_index_z - min_block_index_z + 1;
  CHECK_GE(max_block_index_z, min_block_index_z);

  // Next get a list of all the valid input blocks.
  Index3DSet output_block_set;
  for (const Index3D& block_index : block_indices) {
    if (block_index.z() >= min_block_index_z &&
        block_index.z() <= max_block_index_z) {
      output_block_set.insert(
          Index3D(block_index.x(), block_index.y(), output_block_index_z));
    }
  }

  // Resize everything to the final size.
  size_t num_blocks = output_block_set.size();
  block_indices_host_.resize(num_blocks);
  updated_indices_device_.resize(num_blocks);
  to_clear_indices_device_.resize(num_blocks);

  if (num_blocks == 0) {
    return;
  }

  // Reset the counters.
  if (updated_counter_device_ == nullptr || updated_counter_host_ == nullptr) {
    updated_counter_device_ = make_unified<int>(MemoryType::kDevice);
    updated_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  updated_counter_device_.setZero();
  if (cleared_counter_device_ == nullptr || cleared_counter_host_ == nullptr) {
    cleared_counter_device_ = make_unified<int>(MemoryType::kDevice);
    cleared_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  cleared_counter_device_.setZero();

  // Convert to vector and allocate output blocks.
  size_t i = 0;
  for (const Index3D& block_index : output_block_set) {
    // This is for the output block, which we allocate along the way.
    block_indices_host_[i] = block_index;
    esdf_layer->allocateBlockAtIndex(block_index);

    // Go through all the relevant input pointers:
    Index3D input_block_index = block_index;
    i++;
  }
  block_indices_device_ = block_indices_host_;

  using BlockType = typename LayerType::BlockType;

  // Get the GPU hash of both the TSDF and the ESDF.
  GPULayerView<EsdfBlock> esdf_layer_view = esdf_layer->getGpuLayerView();
  GPULayerView<BlockType> tsdf_layer_view = input_layer.getGpuLayerView();

  // Get the marking functions for this layer type
  auto site_functor = getSiteFunctor(input_layer);

  // Figure out the size of the kernel.
  int dim_block = num_blocks;
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, num_vertical_layers);
  // Call the kernel!
  markSitesInSliceCombinedKernel<BlockType>
      <<<dim_block, dim_threads, 0, cuda_stream_>>>(
          num_blocks, block_indices_device_.data(),
          tsdf_layer_view.getHash().impl_, esdf_layer_view.getHash().impl_,
          site_functor, max_squared_distance_vox, output_voxel_index.z(), min_z,
          max_z, input_layer.block_size(), updated_blocks->data(),
          updated_counter_device_.get(), cleared_blocks->data(),
          cleared_counter_device_.get());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));
  checkCudaErrors(hipPeekAtLastError());

  timing::Timer pack_out_timer("esdf/integrate/mark_sites/pack_out");
  updated_counter_device_.copyTo(updated_counter_host_);
  cleared_counter_device_.copyTo(cleared_counter_host_);

  updated_blocks->resize(*updated_counter_host_);
  cleared_blocks->resize(*cleared_counter_host_);
  pack_out_timer.Stop();
}

__host__ __device__ void getDirectionAndVoxelIndicesFromThread(
    dim3 thread_index, Index3D* block_direction, Index3D* voxel_index,
    Index3D* neighbor_voxel_index, int* axis, int* direction) {
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  *block_direction = Index3D::Zero();
  // Thread index z is used for the neighbor number. x & y are actual voxel
  // indices.
  (*block_direction)(thread_index.z / 2) = thread_index.z % 2 ? -1 : 1;
  // This is to to make it easier to pick voxel directions.
  *axis = thread_index.z / 2;
  *direction = thread_index.z % 2 ? -1 : 1;

  // Fill in the axes.
  if (*axis == 0) {
    *voxel_index << 0, thread_index.x, thread_index.y;
  } else if (*axis == 1) {
    *voxel_index << thread_index.x, 0, thread_index.y;
  } else if (*axis == 2) {
    *voxel_index << thread_index.x, thread_index.y, 0;
  }
  *neighbor_voxel_index = *voxel_index;
  // If we're looking backwards...
  if (*direction < 0) {
    (*voxel_index)(*axis) = 0;
    (*neighbor_voxel_index)(*axis) = kVoxelsPerSide - 1;
  } else {
    (*voxel_index)(*axis) = kVoxelsPerSide - 1;
    (*neighbor_voxel_index)(*axis) = 0;
  }
}

// Thread size MUST be 8x8x6, 8x8 being the side of the cube, and 6 being the
// number of neighbors considered per block. Block size can be whatever.
__global__ void updateNeighborBandsCombinedKernel(
    int i, int num_blocks, Index3DDeviceHashMapType<EsdfBlock> block_hash,
    float max_squared_distance_vox, Index3D* block_indices,
    Index3D* output_vector, int* updated_size) {
  // For every block in the initial list, look up its neighbors.
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  constexpr int kNumNeighbors = 6;

  // Luckily the direction is the same for all processed blocks by this thread.
  Index3D block_direction, voxel_index, neighbor_voxel_index;
  int axis, direction;
  getDirectionAndVoxelIndicesFromThread(threadIdx, &block_direction,
                                        &voxel_index, &neighbor_voxel_index,
                                        &axis, &direction);

  __shared__ bool block_updated;
  // Allow block size to be whatever.
  __shared__ EsdfBlock* block_ptr;
  EsdfBlock* neighbor_block_ptr = nullptr;
  for (int block_idx = blockIdx.x; block_idx < num_blocks;
       block_idx += gridDim.x) {
    __syncthreads();
    // Get the current block for this... block.
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
      block_ptr = nullptr;
      auto it = block_hash.find(block_indices[block_idx]);
      if (it != block_hash.end()) {
        block_ptr = it->second;
      }
      block_updated = false;
    }
    __syncthreads();
    // This block doesn't exist. Who knows why. This shouldn't happen.
    if (block_ptr == nullptr) {
      continue;
    }

    dim3 specific_thread = threadIdx;
    specific_thread.z = i;
    Index3D block_direction, voxel_index, neighbor_voxel_index;
    int axis, direction;
    getDirectionAndVoxelIndicesFromThread(specific_thread, &block_direction,
                                          &voxel_index, &neighbor_voxel_index,
                                          &axis, &direction);

    // Get the neighbor block for this thread.
    neighbor_block_ptr = nullptr;
    auto it = block_hash.find(block_indices[block_idx] + block_direction);
    if (it != block_hash.end()) {
      neighbor_block_ptr = it->second;
    }
    // Our neighbor doesn't exist. This is fine and normal. Happens to
    // everyone.
    if (neighbor_block_ptr == nullptr) {
      continue;
    }

    bool updated = updateSingleNeighbor(
        block_ptr, voxel_index, neighbor_voxel_index, axis, direction,
        max_squared_distance_vox, neighbor_block_ptr);
    // No bother with atomics.
    if (updated) {
      block_updated = updated;
    }

    __syncthreads();
    if ((threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) &&
        block_updated) {
      //*any_updated = true;
      output_vector[atomicAdd(updated_size, 1)] =
          block_indices[block_idx] + block_direction;
    }
  }
}

template <int kBlockThreads, int kItemsPerThread>
__global__ void sortUniqueKernel(Index3D* indices, int num_indices,
                                 int* num_output_indices) {
  typedef uint64_t IndexHashValue;
  typedef int OriginalIndex;

  typedef hipcub::BlockRadixSort<uint64_t, kBlockThreads, kItemsPerThread,
                              OriginalIndex>
      BlockRadixSortT;
  typedef hipcub::BlockDiscontinuity<IndexHashValue, kBlockThreads>
      BlockDiscontinuityT;
  typedef hipcub::BlockScan<OriginalIndex, kBlockThreads> BlockScanT;

  // Allocate type-safe, repurposable shared memory for collectives
  __shared__ union {
    typename BlockRadixSortT::TempStorage sort;
    typename BlockDiscontinuityT::TempStorage discontinuity;
    typename BlockScanT::TempStorage scan;
  } temp_storage;

  // First we create a values list which is actually the indicies.
  // Obtain this block's segment of consecutive keys (blocked across threads)
  uint64_t thread_keys[kItemsPerThread];
  Index3D thread_values[kItemsPerThread];
  int thread_inds[kItemsPerThread];
  int head_flags[kItemsPerThread];
  int head_indices[kItemsPerThread];
  int thread_offset = threadIdx.x * kItemsPerThread;

  // Fill in the keys from the values.
  // I guess we can just do a for loop. kItemsPerThread should be fairly small.
  Index3DHash index_hash;
  for (int i = 0; i < kItemsPerThread; i++) {
    if (thread_offset + i >= num_indices) {
      // We just pack the key with a large value.
      thread_values[i] = Index3D::Zero();
      thread_keys[i] = SIZE_MAX;
      thread_inds[i] = -1;
    } else {
      thread_values[i] = indices[thread_offset + i];
      thread_keys[i] = index_hash(thread_values[i]);
      thread_inds[i] = thread_offset + i;
    }
  }

  // We then sort the values.
  __syncthreads();
  // Collectively sort the keys
  BlockRadixSortT(temp_storage.sort).Sort(thread_keys, thread_inds);
  __syncthreads();
  // We remove duplicates by find when the discontinuities happen.
  BlockDiscontinuityT(temp_storage.discontinuity)
      .FlagHeads(head_flags, thread_keys, hipcub::Inequality());
  __syncthreads();
  // Get the indices that'll be assigned to the new unique values.
  BlockScanT(temp_storage.scan)
      .InclusiveSum<kItemsPerThread>(head_flags, head_indices);
  __syncthreads();

  // Cool now write only 1 instance of the unique entries to the output.
  for (int i = 0; i < kItemsPerThread; i++) {
    if (thread_offset + i < num_indices) {
      if (head_flags[i] == 1) {
        // Get the proper value out. Cache this for in-place ops next step.
        thread_values[i] = indices[thread_inds[i]];
        atomicMax(num_output_indices, head_indices[i]);
      }
    }
  }
  __syncthreads();

  // Have to do this twice since we do this in-place. Now actually replace
  // the values.
  for (int i = 0; i < kItemsPerThread; i++) {
    if (thread_offset + i < num_indices) {
      if (head_flags[i] == 1) {
        // Get the proper value out.
        indices[head_indices[i] - 1] = thread_values[i];
      }
    }
  }
}

void EsdfIntegrator::sortAndTakeUniqueIndices(
    device_vector<Index3D>* block_indices) {
  if (block_indices->size() == 0) {
    return;
  }
  // Together this should be >> the number of indices
  constexpr int kNumThreads = 128;
  constexpr int kNumItemsPerThread = 4;
  if (block_indices->size() >= kNumThreads * kNumItemsPerThread) {
    LOG(INFO) << "Vector too big to sort. Falling back to thrust.";
    // sort vertices to bring duplicates together
    thrust::sort(thrust::device, block_indices->begin(), block_indices->end(),
                 VectorCompare<Index3D>());

    // Find unique vertices and erase redundancies. The iterator will point to
    // the new last index.
    auto iterator = thrust::unique(thrust::device, block_indices->begin(),
                                   block_indices->end());

    // Figure out the new size.
    size_t new_size = iterator - block_indices->begin();
    block_indices->resize(new_size);
    return;
  }
  if (updated_counter_device_ == nullptr || updated_counter_host_ == nullptr) {
    updated_counter_device_ = make_unified<int>(MemoryType::kDevice);
    updated_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  updated_counter_device_.setZero();

  sortUniqueKernel<kNumThreads, kNumItemsPerThread>
      <<<1, kNumThreads, 0, cuda_stream_>>>(block_indices->data(),
                                            block_indices->size(),
                                            updated_counter_device_.get());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));

  updated_counter_device_.copyTo(updated_counter_host_);
  block_indices->resize(*updated_counter_host_);
}

// Combined functions.
void EsdfIntegrator::updateNeighborBandsCombined(
    device_vector<Index3D>* block_indices, EsdfLayer* esdf_layer,
    float max_squared_distance_vox,
    device_vector<Index3D>* updated_block_indices) {
  if (block_indices->empty()) {
    return;
  }
  timing::Timer sweep_timer("esdf/integrate/compute/neighbor_bands");

  // This function just copies neighbors across block boundaries.
  constexpr int kNumNeighbors = 6;
  constexpr int kUpdatedBlockMultiple = kNumNeighbors;
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;

  updated_block_indices->resize(block_indices->size() * kUpdatedBlockMultiple);
  updated_block_indices->setZero();

  // Create an output variable.
  if (updated_counter_device_ == nullptr || updated_counter_host_ == nullptr) {
    updated_counter_device_ = make_unified<int>(MemoryType::kDevice);
    updated_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  updated_counter_device_.setZero();

  timing::Timer gpu_view("esdf/integrate/compute/neighbor_bands/gpu_view");
  GPULayerView<EsdfBlock> gpu_layer_view = esdf_layer->getGpuLayerView();
  gpu_view.Stop();

  // Call the kernel.
  int dim_block = block_indices->size();
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, 1);
  for (int i = 0; i < kNumNeighbors; i++) {
    updateNeighborBandsCombinedKernel<<<dim_block, dim_threads, 0,
                                        cuda_stream_>>>(
        i, block_indices->size(), gpu_layer_view.getHash().impl_,
        max_squared_distance_vox, block_indices->data(),
        updated_block_indices->data(), updated_counter_device_.get());
  }
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));

  updated_counter_device_.copyTo(updated_counter_host_);
  updated_block_indices->resize(*updated_counter_host_);

  if (*updated_counter_host_ == 0) {
    return;
  }

  timing::Timer copy_out_timer(
      "esdf/integrate/compute/neighbor_bands/copy_out");
  sortAndTakeUniqueIndices(updated_block_indices);
}

/// Thread size MUST be 8x8xN (where N is a number of blocks up to ???), block
/// size can be anything.
__global__ void sweepBlockBandCombinedKernel(
    int num_blocks, Index3DDeviceHashMapType<EsdfBlock> block_hash,
    float max_squared_distance_vox, Index3D* block_indices) {
  // We go one axis at a time, syncing threads in between.
  dim3 thread_index = threadIdx;
  thread_index.z = 0;

  __shared__ EsdfBlock* esdf_block;

  for (int block_idx = blockIdx.x * blockDim.z + threadIdx.z;
       block_idx < num_blocks; block_idx += gridDim.x * blockDim.z) {
    // For simplicity we have to have the same number of blocks in the CUDA
    // kernel call as we have actual blocks.
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      esdf_block = nullptr;
      auto it = block_hash.find(block_indices[block_idx]);
      if (it != block_hash.end()) {
        esdf_block = it->second;
      }
    }
    __syncthreads();
    // This block doesn't exist. Who knows why. This shouldn't happen.
    if (esdf_block == nullptr) {
      continue;
    }
    Index3D voxel_index(0, thread_index.x, thread_index.y);

    // X axis done.
    sweepSingleBand(voxel_index, 0, max_squared_distance_vox, esdf_block);
    __syncthreads();

    // Y axis done.
    voxel_index << thread_index.x, 0, thread_index.y;
    sweepSingleBand(voxel_index, 1, max_squared_distance_vox, esdf_block);
    __syncthreads();

    // Z axis done.
    voxel_index << thread_index.x, thread_index.y, 0;
    sweepSingleBand(voxel_index, 2, max_squared_distance_vox, esdf_block);
    __syncthreads();
  }
}

void EsdfIntegrator::sweepBlockBandCombined(
    device_vector<Index3D>* block_indices, EsdfLayer* esdf_layer,
    float max_squared_distance_vox) {
  if (block_indices->empty()) {
    return;
  }
  timing::Timer sweep_timer("esdf/integrate/compute/sweep");

  // Caching.
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const int num_blocks = block_indices->size();

  GPULayerView<EsdfBlock> gpu_layer_view = esdf_layer->getGpuLayerView();

  // Call the kernel.
  // We do 2-dimensional sweeps in this kernel. Each thread does 3 sweeps.
  // We do 1 blocks at a time because it's faster.
  constexpr int kNumBlocksPerCudaBlock = 1;
  int dim_block = std::max(
      static_cast<int>(
          std::ceil(num_blocks / static_cast<float>(kNumBlocksPerCudaBlock))),
      1);
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, kNumBlocksPerCudaBlock);
  sweepBlockBandCombinedKernel<<<dim_block, dim_threads, 0, cuda_stream_>>>(
      block_indices->size(), gpu_layer_view.getHash().impl_,
      max_squared_distance_vox, block_indices->data());
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));
  checkCudaErrors(hipPeekAtLastError());
}

void EsdfIntegrator::computeEsdfCombined(
    const device_vector<Index3D>& blocks_with_sites, EsdfLayer* esdf_layer) {
  CHECK_NOTNULL(esdf_layer);

  if (blocks_with_sites.size() == 0) {
    return;
  }
  // Cache everything.
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const float voxel_size = esdf_layer->block_size() / kVoxelsPerSide;
  const float max_distance_vox = max_distance_m_ / voxel_size;
  const float max_squared_distance_vox = max_distance_vox * max_distance_vox;

  // First we go over all of the blocks with sites.
  // We compute all the proximal sites inside the block first.
  block_indices_device_ = blocks_with_sites;
  sweepBlockBandCombined(&block_indices_device_, esdf_layer,
                         max_squared_distance_vox);

  int i = 0;
  while (!block_indices_device_.empty()) {
    updateNeighborBandsCombined(&block_indices_device_, esdf_layer,
                                max_squared_distance_vox,
                                &updated_indices_device_);
    sweepBlockBandCombined(&updated_indices_device_, esdf_layer,
                           max_squared_distance_vox);

    timing::Timer swap_timer("esdf/integrate/compute/swap");
    std::swap(block_indices_device_, updated_indices_device_);
    swap_timer.Stop();
  }
}

__device__ void getBlockAndVoxelIndexFromOffset(const Index3D& block_index,
                                                const Index3D& voxel_index,
                                                const Index3D& voxel_offset,
                                                Index3D* neighbor_block_index,
                                                Index3D* neighbor_voxel_index) {
  // For each axis we have to get the mod and div to get the block index and
  // voxel index.
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;

  for (int i = 0; i < 3; i++) {
    (*neighbor_block_index)(i) =
        block_index(i) + voxel_offset(i) / kVoxelsPerSide;
    (*neighbor_voxel_index)(i) =
        voxel_index(i) + voxel_offset(i) % kVoxelsPerSide;
    if ((*neighbor_voxel_index)(i) >= kVoxelsPerSide) {
      (*neighbor_voxel_index)(i) -= kVoxelsPerSide;
      (*neighbor_block_index)(i)++;
    } else if ((*neighbor_voxel_index)(i) < 0) {
      (*neighbor_voxel_index)(i) += kVoxelsPerSide;
      (*neighbor_block_index)(i)--;
    }
  }
}

__global__ void clearAllInvalidKernel(
    Index3D* block_indices, Index3DDeviceHashMapType<EsdfBlock> block_hash,
    float max_squared_distance_vox, Index3D* output_vector, int* updated_size) {
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  __shared__ int block_updated;
  // Allow block size to be whatever.
  __shared__ EsdfBlock* block_ptr;
  // Get the current block for this... block.
  __shared__ Index3D block_index;
  Index3D voxel_index = Index3D(threadIdx.x, threadIdx.y, threadIdx.z);
  if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
    block_ptr = nullptr;
    block_index = block_indices[blockIdx.x];
    auto it = block_hash.find(block_index);
    if (it != block_hash.end()) {
      block_ptr = it->second;
    }
    block_updated = false;
  }
  __syncthreads();
  // This block doesn't exist. Who knows why. This shouldn't happen.
  if (block_ptr == nullptr) {
    return;
  }

  // Now for our specific voxel we should look up its parent and see if it's
  // still there.
  EsdfVoxel* esdf_voxel =
      &block_ptr->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];

  if (esdf_voxel->observed && !esdf_voxel->is_site &&
      esdf_voxel->parent_direction != Index3D::Zero()) {
    Index3D neighbor_block_index, neighbor_voxel_index;
    getBlockAndVoxelIndexFromOffset(
        block_index, voxel_index, esdf_voxel->parent_direction,
        &neighbor_block_index, &neighbor_voxel_index);

    EsdfVoxel* neighbor_voxel = nullptr;
    if (neighbor_block_index == block_index) {
      neighbor_voxel =
          &block_ptr->voxels[neighbor_voxel_index.x()][neighbor_voxel_index.y()]
                            [neighbor_voxel_index.z()];
    } else {
      // Get the neighboring block.
      auto it = block_hash.find(neighbor_block_index);
      if (it != block_hash.end()) {
        neighbor_voxel =
            &it->second
                 ->voxels[neighbor_voxel_index.x()][neighbor_voxel_index.y()]
                         [neighbor_voxel_index.z()];
      }
    }
    if (neighbor_voxel == nullptr || !neighbor_voxel->is_site) {
      // Clear this voxel.
      esdf_voxel->parent_direction.setZero();
      esdf_voxel->squared_distance_vox = max_squared_distance_vox;
      block_updated = true;
    }
  }
  __syncthreads();
  if ((threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) &&
      block_updated) {
    output_vector[atomicAdd(updated_size, 1)] = block_index;
  }
}

void EsdfIntegrator::clearAllInvalid(
    const std::vector<Index3D>& blocks_to_clear, EsdfLayer* esdf_layer,
    device_vector<Index3D>* updated_blocks) {
  if (blocks_to_clear.size() == 0) {
    return;
  }

  // TODO: start out just getting all the blocks in the whole map.
  // Then replace with blocks within a radius of the cleared blocks.
  constexpr int kVoxelsPerSide = VoxelBlock<bool>::kVoxelsPerSide;
  const float max_distance_vox = max_distance_m_ / esdf_layer->voxel_size();
  const float max_squared_distance_vox = max_distance_vox * max_distance_vox;

  timing::Timer get_blocks_timer("esdf/integrate/clear/get_blocks");

  temp_indices_host_ = getBlocksWithinRadiusOfAABB(
      esdf_layer->getAllBlockIndices(), esdf_layer->block_size(),
      getAABBOfBlocks(esdf_layer->block_size(), blocks_to_clear),
      max_distance_m_);
  get_blocks_timer.Stop();
  temp_indices_device_ = temp_indices_host_;

  // Get the hash map of the whole ESDF map.
  GPULayerView<EsdfBlock> gpu_layer_view = esdf_layer->getGpuLayerView();

  // Create an output variable.
  if (updated_counter_device_ == nullptr || updated_counter_host_ == nullptr) {
    updated_counter_device_ = make_unified<int>(MemoryType::kDevice);
    updated_counter_host_ = make_unified<int>(MemoryType::kHost);
  }
  updated_counter_device_.setZero();

  // Make sure we have enough space if EVERYTHING had to be cleared.
  updated_blocks->resize(temp_indices_device_.size());

  // Call a kernel.
  dim3 dim_threads(kVoxelsPerSide, kVoxelsPerSide, kVoxelsPerSide);
  if (temp_indices_device_.size() > 0) {
    clearAllInvalidKernel<<<temp_indices_device_.size(), dim_threads, 0,
                            cuda_stream_>>>(
        temp_indices_device_.data(), gpu_layer_view.getHash().impl_,
        max_squared_distance_vox, updated_blocks->data(),
        updated_counter_device_.get());
    checkCudaErrors(hipStreamSynchronize(cuda_stream_));
    checkCudaErrors(hipPeekAtLastError());

    // Pack out the updated blocks.
    updated_counter_device_.copyTo(updated_counter_host_);
    updated_blocks->resize(*updated_counter_host_);
  } else {
    updated_blocks->resize(0);
  }
}

// ============================ Output conversions =============================
__global__ void populateSliceFromLayerKernel(
    Index3DDeviceHashMapType<EsdfBlock> block_hash, AxisAlignedBoundingBox aabb,
    float block_size, float* image, int rows, int cols, float z_slice_height,
    float resolution, float unobserved_value, bool image_coordinates) {
  const float voxel_size = block_size / EsdfBlock::kVoxelsPerSide;
  const int pixel_col = blockIdx.x * blockDim.x + threadIdx.x;
  int pixel_row = blockIdx.y * blockDim.y + threadIdx.y;

  if (pixel_col >= cols || pixel_row >= rows) {
    return;
  }

  // Figure out where this pixel should map to.
  Vector3f voxel_position(aabb.min().x() + resolution * pixel_col,
                          aabb.min().y() + resolution * pixel_row,
                          z_slice_height);

  Index3D block_index, voxel_index;

  getBlockAndVoxelIndexFromPositionInLayer(block_size, voxel_position,
                                           &block_index, &voxel_index);

  // If using image coordinates (rather than matrix coordinates), the row
  // numbers are flipped. In image coordinates, upper-left is 0,0, in matrix
  // coordinates, it's lower-left.  Flip the row counter in a corresponding way.
  if (image_coordinates) {
    pixel_row = rows - pixel_row - 1;
  }

  // Get the relevant block.
  EsdfBlock* block_ptr = nullptr;
  auto it = block_hash.find(block_index);
  if (it != block_hash.end()) {
    block_ptr = it->second;
  } else {
    image::access(pixel_row, pixel_col, cols, image) = unobserved_value;
    return;
  }

  // Get the relevant pixel.
  const EsdfVoxel* voxel =
      &block_ptr->voxels[voxel_index.x()][voxel_index.y()][voxel_index.z()];
  float distance = unobserved_value;
  if (voxel->observed) {
    distance = voxel_size * std::sqrt(voxel->squared_distance_vox);
    if (voxel->is_inside) {
      distance = -distance;
    }
  }
  image::access(pixel_row, pixel_col, cols, image) = distance;
}

void EsdfIntegrator::populateSliceFromLayer(
    const EsdfLayer& layer, const AxisAlignedBoundingBox& aabb,
    float z_slice_height, float resolution, float unobserved_value,
    bool image_coordinates, Image<float>* image) {
  if (image->numel() <= 0) {
    return;
  }
  const float voxel_size = layer.voxel_size();

  if (cuda_stream_ == nullptr) {
    checkCudaErrors(hipStreamCreate(&cuda_stream_));
  }

  // Create a GPU hash of the ESDF.
  GPULayerView<EsdfBlock> gpu_layer_view = layer.getGpuLayerView();

  // Pass in the GPU hash and AABB and let the kernel figure it out.
  constexpr int kThreadDim = 16;
  const int rounded_rows = static_cast<int>(
      std::ceil(image->rows() / static_cast<float>(kThreadDim)));
  const int rounded_cols = static_cast<int>(
      std::ceil(image->cols() / static_cast<float>(kThreadDim)));
  dim3 block_dim(rounded_cols, rounded_rows);
  dim3 thread_dim(kThreadDim, kThreadDim);

  populateSliceFromLayerKernel<<<block_dim, thread_dim, 0, cuda_stream_>>>(
      gpu_layer_view.getHash().impl_, aabb, layer.block_size(),
      image->dataPtr(), image->rows(), image->cols(), z_slice_height,
      resolution, unobserved_value, image_coordinates);
  checkCudaErrors(hipStreamSynchronize(cuda_stream_));
  checkCudaErrors(hipPeekAtLastError());
}

void EsdfIntegrator::convertLayerSliceToDistanceImage(
    const EsdfLayer& layer, float unobserved_value, float slice_height,
    bool image_coordinates, Image<float>* output_image,
    AxisAlignedBoundingBox* aabb) {
  // Just call the other version.
  const float block_size = layer.block_size();

  std::vector<Index3D> block_indices = layer.getAllBlockIndices();
  AxisAlignedBoundingBox aabb_all;
  aabb_all.setEmpty();

  // Figure out the index of the desired height.
  Index3D desired_z_block_index;
  Index3D desired_z_voxel_index;
  getBlockAndVoxelIndexFromPositionInLayer(
      block_size, Vector3f(0.0f, 0.0f, slice_height), &desired_z_block_index,
      &desired_z_voxel_index);

  for (const Index3D& block_index : block_indices) {
    // Skip all other heights of block.
    if (block_index.z() != desired_z_block_index.z()) {
      continue;
    }

    // Extend the AABB by the dimensions of this block.
    aabb_all.extend(getAABBOfBlock(block_size, block_index));
  }

  // Call the function that actually creates the output image.
  convertLayerSliceToDistanceImage(layer, unobserved_value, slice_height,
                                   image_coordinates, aabb_all, output_image);

  // Output.
  *aabb = aabb_all;
}

void EsdfIntegrator::convertLayerSliceToDistanceImage(
    const EsdfLayer& layer, float unobserved_value, float slice_height,
    bool image_coordinates, const AxisAlignedBoundingBox& aabb,
    Image<float>* output_image) {
  const float block_size = layer.block_size();
  constexpr int kVoxelsPerSide = VoxelBlock<EsdfVoxel>::kVoxelsPerSide;
  const float voxel_size = block_size / kVoxelsPerSide;

  Vector3f bounding_size = aabb.sizes();
  // Width = cols, height = rows
  int width = static_cast<int>(std::ceil(bounding_size.x() / voxel_size));
  int height = static_cast<int>(std::ceil(bounding_size.y() / voxel_size));

  // Create an image on the device to fit the aabb.
  Image<float> image(height, width, MemoryType::kDevice);

  // Fill in the float image.
  populateSliceFromLayer(layer, aabb, slice_height, voxel_size,
                         unobserved_value, image_coordinates, &image);

  *output_image = std::move(image);
  checkCudaErrors(hipPeekAtLastError());
}

}  // namespace nvblox
